#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "utilCuda.cuh"

__global__ void probabilities_calculation(double *pheromone, double *probabilities, double sum, bool *visited, double *distances, int alpha, int beta)
{
    int i = threadIdx.x;

    if (visited[i] || distances[i] == 0)
        probabilities[i] = 0;
    else
        probabilities[i] = pow(pheromone[i], alpha) * pow(1 / distances[i], beta) / sum;
}

__global__ void copy_vector(int *vector1, int *vector2)
{
    int i = threadIdx.x;
    vector1[i] = vector2[i];
}
